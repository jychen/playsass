#include "hip/hip_runtime.h"

__global__ void vote_any_kernel(int *a, int b)
{
    *a = __any_sync(0xFFFFFFFF, b >> threadIdx.x);
}

__global__ void ballot_sync_kernel(int *a, int b)
{
    *a = __ballot_sync(0xFFFFFFFF, b >> threadIdx.x);
}

int main(int argc, char **argv)
{
    return 0;
}

