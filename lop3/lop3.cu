#include "hip/hip_runtime.h"

__global__ void and_kernel(int *a, int b)
{
    a[0] = a[0] & b;
}

__global__ void xor_kernel(int *a, int b)
{
    a[0] ^= b;
}

__global__ void or_kernel(int *a, int b)
{
    a[0] |= b;
}

__global__ void and3_kernel(int *a, int b, int c)
{
    a[0] &= b & c;
}

__global__ void xor3_kernel(int *a, int b, int c)
{
    a[0] ^= b ^ c;
}

__global__ void or3_kernel(int *a, int b, int c)
{
    a[0] |= b | c;
}

__global__ void andnot_kernel(int *a, int b)
{
    a[0] = ~a[0] & b;
}

__global__ void xorandnot_kernel(int *a, int b, int c)
{
    a[0] = a[0] ^ (~b & c);
}

int main(int argc, char **argv)
{
    return 0;
}

