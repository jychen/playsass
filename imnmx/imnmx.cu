#include "hip/hip_runtime.h"

__global__ void max_kernel(int *a, int b)
{
    *a = max(*a, b);
}

__global__ void min_kernel(int *a, int b)
{
    *a = min(*a, b);
}

int main(int argc, char **argv)
{
    return 0;
}

