#include "hip/hip_runtime.h"

__global__ void shr_const_kernel(uint32_t *a)
{
    a[0] >>= 3;
}

__global__ void shr_var_kernel(uint32_t *a, int b)
{
    a[0] >>= b;
}

__global__ void shl_const_kernel(uint32_t *a)
{
    a[0] <<= 3;
}

__global__ void shl_var_kernel(uint32_t *a, int b)
{
    a[0] <<= b;
}

int main(int argc, char **argv)
{
    shr_const_kernel<<<1, 1>>>(NULL);
    shr_var_kernel<<<1, 1>>>(NULL, 20);
    shl_const_kernel<<<1, 1>>>(NULL);
    shl_var_kernel<<<1, 1>>>(NULL, 20);
    return 0;
}

