#include "hip/hip_runtime.h"

__global__ void madlo_kernel(uint32_t *a)
{
    a[0] = a[0] * a[1];
}

__global__ void madwide_kernel(uint32_t *a)
{
    uint32_t b = (uint64_t)a[0] * (uint64_t)a[1] >> 32;
    a[0] = b;
}

__global__ void mad_ptx_kernel(uint32_t *a, uint32_t *b)
{
    uint64_t result;
    asm volatile("mad.wide.u32 %0, %1, %2, %3 ;\n" : "=l"(result) : "r"(a[0]), "r"(a[1]), "l"(*(uint64_t *)a));
    b[0] = result & 0xFFFFFFFF;
    b[1] = result >> 32;
}

__global__ void mad_cc_ptx_kernel(uint32_t *a, uint32_t *b)
{
    uint32_t result[5] = {0};
    asm volatile("mad.lo.cc.u32 %0, %5, %6, %7 ;\n"
                 "madc.hi.cc.u32 %1, %5, %6, %8 ;\n" 
                 "madc.lo.cc.u32 %2, %9, %10, %11;\n"
                 "madc.hi.cc.u32 %3, %9, %10, %12;\n"
                 "addc.cc.u32 %4, %13, %14 ;\n"
                 : "=r"(result[0]), "=r"(result[1]), "=r"(result[2]), "=r"(result[3]), "=r"(result[4])
                 : "r"(a[0]), "r"(b[0]), "r"(0), "r"(0), "r"(a[1]), "r"(b[1]), "r"(0), "r"(0), "r"(0), "r"(0));
    for (uint32_t i = 0; i < 5; i++) {
        b[i] = result[i];
    }
}


int main(int argc, char **argv)
{
    madlo_kernel<<<1, 1>>>(NULL);
    madwide_kernel<<<1, 1>>>(NULL);
    mad_ptx_kernel<<<1, 1>>>(NULL, NULL);
    return 0;
}

/**
 * IMAD d, a, b, c ;                        d = a * b + c
 * IMAD.WIDE.U32 d, P, a, b, c ;            d = a * b + c, P = carryOut
 * IMAD.WIDE.U32.X d, a, b, c, P ;          d = a * b + c + P
 * IMAD.WIDE.U32.X d, P1, a, b, c, P2 ;     d = a * b + c + P2, P1 = carryOut
*/