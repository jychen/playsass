#include "hip/hip_runtime.h"

#define DIVISOR 19

__global__ void divide_by_const(uint32_t *a)
{
    a[0] = a[0] / DIVISOR;
}

__global__ void divide_by_var(uint32_t *a, uint32_t b)
{
    a[0] = a[0] / b;
}

int main(int argc, char **argv)
{
    uint32_t var = atoi(argv[1]);
    divide_by_const<<<1, 1>>>(NULL);
    divide_by_var<<<1, 1>>>(NULL, var);
    return 0;
}