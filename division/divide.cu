#include "hip/hip_runtime.h"

#define DIVISOR 3256

__global__ void divide_by_const(uint32_t *a)
{
    a[0] = a[0] / DIVISOR;
}

__global__ void divide_by_const_64(uint64_t *a)
{
    a[0] = a[0] / DIVISOR;
}

__global__ void mod_by_const_64(uint64_t *a)
{
    a[0] = a[0] % DIVISOR;
}

__global__ void divide_by_var(uint32_t *a, uint32_t b)
{
    a[0] = a[0] / b;
}

__global__ void divide_by_var_64(uint64_t *a, uint64_t b)
{
    a[0] = a[0] / b;
}

int main(int argc, char **argv)
{
    uint32_t var = atoi(argv[1]);
    divide_by_const<<<1, 1>>>(NULL);
    divide_by_var<<<1, 1>>>(NULL, var);
    return 0;
}