#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

__global__ void syncthreads_kernel()
{
    __syncthreads();
}

__global__ void threadfence_kernel()
{
    __threadfence();
}

__global__ void groupsync_kernel(int *a, int b)
{
    auto g = cooperative_groups::this_grid();
    g.sync();
    if (threadIdx.x == 0) *a = b;
}

__global__ void warpsync_kernel(int *a)
{
    auto warp = cooperative_groups::coalesced_threads();
    warp.sync();
    if (threadIdx.x == 0) *a = 100;
}

int main(int argc, char **argv)
{
    return 0;
}

