#include "hip/hip_runtime.h"

__global__ void lea_kernel(uint32_t *a, uint32_t off1, uint32_t off2)
{
    a[0] = a[(off1 << 8) + off2];
}


int main(int argc, char **argv)
{
    uint32_t off1 = atoi(argv[1]);
    uint32_t off2 = atoi(argv[2]);
    lea_kernel<<<1, 1>>>(NULL, off1, off2);
    return 0;
}


/**
 * LEA d, a, b, c ;                             d = (a << c) + b
 * LEA dst, carry, index, base, shift;          (dst, carry) = base + lo(index << shift)
 * LEA.HI dst, index, baseHi, shift, carry;     dst = baseHi + hi(index << shift) + carry
*/